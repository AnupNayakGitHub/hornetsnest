#include "StaticBreadthFirstSearch/TopDown++.cuh"

int main(int argc, char* argv[]) {
    using namespace custinger;
    using namespace custinger_alg;
    hipSetDevice(1);

    graph::GraphStd<vid_t, eoff_t> graph;
    graph.read(argv[1]);

    cuStingerInit custinger_init(graph.nV(), graph.nE(), graph.out_offsets(),
                                 graph.out_edges());

    cuStinger custiger_graph(custinger_init);

    BfsTopDown2 bfs_top_down(custiger_graph);
    bfs_top_down.set_parameters(0);
    bfs_top_down.run();

    auto is_correct = bfs_top_down.validate();
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
    return is_correct;
}
