/**
 * @brief Sparse Matrix-Vector multiplication
 * @file
 */
#include "Static/SpMV/SpMV.cuh"
#include <GraphIO/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
//#include <hip/hip_runtime_api.h> //--profile-from-start off

int main(int argc, char* argv[]) {
    using namespace timer;
    using namespace hornet_alg;

    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.out_offsets_ptr(),
                           graph.out_edges_ptr());

    HornetGPU hornet_matrix(hornet_init);

    SpMV spmv(hornet_matrix);

    Timer<DEVICE> TM;
    TM.start();

    spmv.run();

    TM.stop();
    TM.print("SpMV");

    auto is_correct = spmv.validate();
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
    return is_correct;
}
