/**
 * @brief Breadth-first Search Top-Down test program (C++11 Style APIs)
 * @file
 */
#include "Static/BreadthFirstSearch/TopDown.cuh"
#include <GraphIO/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
//#include <hip/hip_runtime_api.h> //--profile-from-start off

using namespace hornet_alg;

int main(int argc, char* argv[]) {
    using namespace timer;

    graph::GraphStd<vid_t, eoff_t> graph;
    CommandLineParam cmd(graph, argc, argv);

    HornetInit hornet_init(graph.nV(), graph.nE(), graph.out_offsets_ptr(),
                          graph.out_edges_ptr());

    //HornetCSR hornet_graph(hornet_init);
    HornetGPU hornet_graph(hornet_init);

    BfsTopDown2 bfs_top_down(hornet_graph);

    //bfs_top_down.set_parameters(graph.max_out_degree_id());
    bfs_top_down.set_parameters(0);

    Timer<DEVICE> TM;
    TM.start();
    //hipProfilerStart();

    bfs_top_down.run();

    //hipProfilerStop();
    TM.stop();
    TM.print("TopDown");

    auto is_correct = bfs_top_down.validate();
    std::cout << (is_correct ? "\nCorrect <>\n\n" : "\n! Not Correct\n\n");
    return is_correct;
}
