/**
 * @brief
 * @file
 */

#include "HornetAlg.hpp"
#include "Core/LoadBalancing/VertexBased.cuh"
#include "Core/LoadBalancing/ScanBased.cuh"
#include "Core/LoadBalancing/BinarySearch.cuh"
#include <Core/GPUCsr/Csr.cuh>
#include <Core/GPUHornet/Hornet.cuh>
#include <GraphIO/GraphStd.hpp>
#include <Util/CommandLineParam.hpp>
#include <hip/hip_runtime_api.h> //--profile-from-start off

#include "Static/TriangleCounting/triangle2.cuh"

using namespace timer;
using namespace hornets_nest;

using HornetGraph = gpu::Hornet<EMPTY, EMPTY>;




int main(int argc, char* argv[]) {

    using namespace graph::structure_prop;
    using namespace graph::parsing_prop;

    graph::GraphStd<vid_t, eoff_t> graph(UNDIRECTED);
    graph.read(argv[1], SORT | PRINT_INFO);
    HornetInit hornet_init(graph.nV(), graph.nE(), graph.csr_out_offsets(),
                           graph.csr_out_edges());

    HornetGraph hornet_graph(hornet_init);
    // hornet_graph.print();
    TriangleCounting tc(hornet_graph);
    tc.init();

    Timer<DEVICE> TM(5);
    hipProfilerStart();
    TM.start();

    tc.run();

    TM.stop();
    hipProfilerStop();
    TM.print("ForAllAdjUnions Time");

    return 0;
}
